#include <sys/time.h>
#include <stdio.h>
//TODO for writing to file, will be deleted
#include <stdlib.h>
//TODO: could include later
//#include <device_launch_parameters.h>
#include <hip/hip_runtime.h>
//#include "../inc/helper_cuda.h"
#define NUM_STREAMS 8 
double timeStampB;
double timeStampC;
double timeStampD;
// time stamp function in seconds
double getTimeStamp() {
 struct timeval tv ;
 gettimeofday( &tv, NULL ) ;
 return (double) tv.tv_usec/1000000 + tv.tv_sec ;
}
void myCallBackB(hipStream_t stream,hipError_t status, void*  userData ){
 timeStampB=getTimeStamp();
}
void myCallBackC(hipStream_t stream,hipError_t status, void*  userData ){
 timeStampC=getTimeStamp();
}
void myCallBackD(hipStream_t stream,hipError_t status, void*  userData ){
 timeStampD=getTimeStamp();
}
void initDataA(float* data, int nx, int ny){ 
 int i,j;
 for(i = 0; i < nx; i++){
  for(j = 0; j < ny; j++){
   data[i*ny + j] = (float) (i+j)/3.0;
  }
 }
}
void initDataB(float* data, int nx, int ny){ 
 int i,j;
 for(i = 0; i < nx; i++){
  for(j = 0; j < ny; j++){
   data[i*ny + j] = (float)3.14*(i+j);
  }
 }
}
void debugPrint(float* data, int nx, int ny){
 int i,j;
 for(i = 0; i < nx; i++){
  for(j = 0; j < ny; j++){
   printf("%f ",data[i*ny + j]);
  }
  printf("\n");
 }
 printf("\n");
}
// host side matrix addition
void h_addmat(float *A, float *B, float *C, int nx, int ny){
 int i;
 for(i = 0; i < nx*ny; i++){
  C[i] = A[i] + B[i];
 }
}
// device-side matrix addition
__global__ void f_addmat( float *A, float *B, int len/*, int padrow*/){
 // kernel code might look something like this
 // but you may want to pad the matrices and index into them accordingly
 //__shared__ float sA[32][32];
 //__shared__ float sB[32][32];
 //__shared__ float sC[32][32];

 int ix = threadIdx.x;
 int iy = threadIdx.y*blockDim.x + blockIdx.x*blockDim.x*blockDim.y;
 int idx = iy + ix ;
 //int col = idx-padrow*(int)(idx/padrow);
 //if(idx<nx*padrow && col<ny){
 //if(idx<gridDim.x/4*blockDim.x*blockDim.y){
  //int sidx = threadIdx.y*blockDim.x + threadIdx.x;
  //int size = ((nx*ny-idx)<4) ? (nx*ny-idx) : 4;
  //int size=4;
  //if((ny-col)<4){
  // size = ny-col;
  //}
  //if(col<4){
  // size += col;
  //}

  //float tmpA[4];
  //float tmpB[4];
  //memcpy(tmpA,&A[idx],size);
  //memcpy(tmpB,&B[idx],size);
  //for(int j = 0; j < size; j++){
  // tmpB[j] += tmpA[j];
  //}
  //memcpy(&B[idx],tmpB,size);
  //printf("sidx is %d, idx is %d, size is %d\n", sidx, idx, size);
  #pragma unroll
  for(int i = idx; i < len; i+=gridDim.x*blockDim.x*blockDim.y){
   //sA[threadIdx.x][threadIdx.y] = A[i];
   //sB[threadIdx.x][threadIdx.y] = B[i];
   //__syncthreads();
   //sC[threadIdx.x][threadIdx.y] = sA[threadIdx.x][threadIdx.y] + sB[threadIdx.x][threadIdx.y];
   //__syncthreads();
   //C[i] = sC[threadIdx.x][threadIdx.y];
   //printf("index %d\n",i);
   B[i] += A[i];
  }
 //}
}
int main( int argc, char *argv[] ) {
 // get program arguments
 if( argc != 3) {
 printf("Error: wrong number of args\n") ;
 exit(1) ;
 }
 int nx = atoi( argv[1] ) ; // should check validity
 int ny = atoi( argv[2] ) ; // should check validity
 int noElems = nx*ny ;
 int bytes = noElems * sizeof(float) ;
 // but you may want to pad the matrices…
 
 // alloc memory host-side
 float *h_hA = (float *) malloc( bytes ) ;
 float *h_hB = (float *) malloc( bytes ) ;
 float *h_hC = (float *) malloc( bytes ) ; // host result
 //float *h_dC = (float *) malloc( bytes ) ; // gpu result
 float *h_A, *h_B, *h_dC;
 float *d_A, *d_B ;
 hipHostAlloc((void**)&h_A,bytes,hipHostMallocWriteCombined|hipHostMallocMapped);
 hipHostAlloc((void**)&h_B,bytes,hipHostMallocWriteCombined|hipHostMallocMapped);
 hipHostAlloc((void**)&h_dC,bytes,hipHostMallocWriteCombined);
 //cudaHostGetDevicePointer( &d_A, h_A, 0 );
 //cudaHostGetDevicePointer( &d_B, h_B, 0 );
 //cudaHostGetDevicePointer( &d_C, h_dC, 0 );
 // init matrices with random data
 //initData( h_A, noElems ) ; initData( h_B, noElems ) ;
 initDataA(h_A, nx, ny);
 initDataB(h_B, nx, ny);
 initDataA(h_hA, nx, ny);
 initDataB(h_hB, nx, ny);
 // alloc memory dev-side
 hipMalloc( (void **) &d_A, bytes ) ;
 hipMalloc( (void **) &d_B, bytes ) ;
 hipDeviceSetCacheConfig(hipFuncCachePreferL1);
 double timeStampA = getTimeStamp() ;
 //transfer data to dev
 //cudaMemcpy( d_A, h_A, bytes, cudaMemcpyHostToDevice ) ;
 //cudaMemcpy( d_B, h_B, bytes, cudaMemcpyHostToDevice ) ;
 //printf("pA is %d and pB is %d\n",pitchA,pitchB);
 //cudaMemcpy2D( d_A, pitchA, h_A,ny*sizeof(float),ny*sizeof(float),nx,cudaMemcpyHostToDevice ) ;
 //cudaMemcpy2D( d_B, pitchB, h_B,ny*sizeof(float),ny*sizeof(float),nx,cudaMemcpyHostToDevice ) ;
 // note that the transfers would be twice as fast if h_A and h_B
 // matrices are pinned
 //double timeStampB = getTimeStamp() ;
 //double timeStampC = getTimeStamp() ;
 //double timeStampD = getTimeStamp() ;

 // invoke Kernel
 dim3 block( 32, 32 ) ; // you will want to configure this
 //int block = 64;
 //int grid = (noElems + block-1)/block;
 int grid = ((noElems+3)/4/NUM_STREAMS + block.x*block.y-1)/(block.x*block.y);
 //int grid = ((noElems/NUM_STREAMS+3)/4 + block.x*block.y-1)/(block.x*block.y);
 //int grid = (((pitchA/4*nx*sizeof(float))+3)/4 + block.x*block.y-1)/(block.x*block.y);
 //dim3 grid( (nx + block.x-1)/block.x, (ny + block.y-1)/block.y ) ;
 //cudaDeviceProp GPUprop;
 //cudaGetDeviceProperties(&GPUprop,0);
 //printf("sharedmemperblk is %d\n",GPUprop.sharedMemPerBlock);
 //printf("maxgridsize x is %d\n",GPUprop.maxGridSize[0]);
 //printf("noelems is %d\n",noElems);
 //printf("prev num is %d\n",noElems/NUM_STREAMS);
 //printf("align num is %d\n",noElems/NUM_STREAMS-(noElems/NUM_STREAMS)%8);
 int align_idx = noElems/NUM_STREAMS-(noElems/NUM_STREAMS)%8;
 //printf("grid is %d\n",grid);
 //printf("gridx is %d and grid y is %d\n",grid.x,grid.y);

 //f_addmat<<<grid, block>>>( d_A, d_B, nx, ny/*, pitchA/(sizeof(float))*/ ) ;
 //cudaDeviceSynchronize() ;

 hipStream_t stream[NUM_STREAMS+1];
 for (int i = 1; i < NUM_STREAMS+1; i++){
  hipStreamCreate(&(stream[i]));
 }
 int i;
 for(i = 1; i < NUM_STREAMS; i++){
  hipMemcpyAsync(&d_A[(i-1)*align_idx],&h_A[(i-1)*align_idx],align_idx*sizeof(float),hipMemcpyHostToDevice,stream[i]);
  hipMemcpyAsync(&d_B[(i-1)*align_idx],&h_B[(i-1)*align_idx],align_idx*sizeof(float),hipMemcpyHostToDevice,stream[i]);
  hipStreamAddCallback(stream[i],myCallBackB,(void*)&i,0);
  //printf("index is %d, num is %d\n",(i-1)*nx*ny/NUM_STREAMS,nx*ny/NUM_STREAMS );
  f_addmat<<<grid, block, 0, stream[i]>>>( d_A+(i-1)*align_idx, d_B+(i-1)*align_idx,align_idx) ;
  hipStreamAddCallback(stream[i],myCallBackC,(void*)&i,0);
  hipMemcpyAsync(&h_dC[(i-1)*align_idx],&d_B[(i-1)*align_idx],align_idx*sizeof(float),hipMemcpyDeviceToHost,stream[i]);
  hipStreamAddCallback(stream[i],myCallBackD,(void*)&i,0);
 }
 grid =((noElems-(NUM_STREAMS-1)*align_idx+3)/4+ block.x*block.y-1)/(block.x*block.y);
 //printf("grid final is %d\n",grid);
 //printf("index is %d, num is %d\n",(NUM_STREAMS-1)*nx*ny/NUM_STREAMS,nx*ny-(NUM_STREAMS-1)*nx*ny/NUM_STREAMS);
 hipMemcpyAsync(&d_A[(NUM_STREAMS-1)*align_idx],&h_A[(NUM_STREAMS-1)*align_idx],(noElems-(NUM_STREAMS-1)*align_idx)*sizeof(float),hipMemcpyHostToDevice,stream[NUM_STREAMS]);
 hipMemcpyAsync(&d_B[(NUM_STREAMS-1)*align_idx],&h_B[(NUM_STREAMS-1)*align_idx],(noElems-(NUM_STREAMS-1)*align_idx)*sizeof(float),hipMemcpyHostToDevice,stream[NUM_STREAMS]);
 hipStreamAddCallback(stream[i],myCallBackB,(void*)&i,0);
 f_addmat<<<grid, block, 0, stream[NUM_STREAMS]>>>( d_A+(NUM_STREAMS-1)*align_idx, d_B+(NUM_STREAMS-1)*align_idx,noElems-(NUM_STREAMS-1)*align_idx) ;
 hipStreamAddCallback(stream[i],myCallBackC,(void*)&i,0);
 hipMemcpyAsync(&h_dC[(NUM_STREAMS-1)*align_idx],&d_B[(NUM_STREAMS-1)*align_idx],(noElems-(NUM_STREAMS-1)*align_idx)*sizeof(float),hipMemcpyDeviceToHost,stream[NUM_STREAMS]);
 hipStreamAddCallback(stream[i],myCallBackD,(void*)&i,0);
 for(int i = 1; i < NUM_STREAMS+1; i++){
  hipStreamSynchronize(stream[i]);
 }
 //f_addmat<<<grid, block>>>( d_A, d_B, nx*ny/*, pitchA/(sizeof(float))*/ ) ;
 //cudaDeviceSynchronize() ;
 //copy data back
 //cudaMemcpy( h_dC, d_B, bytes, cudaMemcpyDeviceToHost ) ;
 //cudaMemcpy2D( h_ddC, pitchB, d_B,ny*sizeof(float),ny*sizeof(float),nx,cudaMemcpyDeviceToHost ) ;

 // check result
 h_addmat( h_hA, h_hB, h_hC, nx, ny ) ;

 //for(int i = 0; i < nx; i++){
 // for(int j = 0; j < pitchC/4; j++){
 //  if(j<ny){
 //   h_dC[i*ny+j] = h_ddC[i*pitchC/4+j];
 //  }
 // }
 //} 
 // print out results
 if(!memcmp(h_hC,h_dC,nx*ny*sizeof(float))){
 //if(1){
  //debugPrint(h_hC, nx, ny);
  //debugPrint(h_dC, nx, ny);
  FILE* fptr;
  fptr = fopen("time.log","a");
  fprintf(fptr,"%dX%d %.6f %.6f %.6f %.6f\n", nx, ny, timeStampD-timeStampA, timeStampB-timeStampA, timeStampC-timeStampB, timeStampD-timeStampC);
  fclose(fptr);
  printf("%.6f %.6f %.6f %.6f\n", timeStampD-timeStampA, timeStampB-timeStampA, timeStampC-timeStampB, timeStampD-timeStampC);
 }else{
  //debugPrint(h_hC, nx, ny);
  //debugPrint(h_dC, nx, ny);
  printf("Error: Results not matched.\n");
 }
 // free GPU resources
 hipHostFree( h_A ) ; hipHostFree( h_B ) ; hipHostFree( h_dC ) ;
 hipDeviceReset() ;
}
