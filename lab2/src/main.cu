#include <sys/time.h>
#include <stdio.h>
#include<math.h>
//TODO for writing to file, will be deleted
#include <stdlib.h>
//TODO: could include later
//#include <device_launch_parameters.h>
#include <hip/hip_runtime.h>
//#include "../inc/helper_cuda.h"

// time stamp function in ms
double getTimeStamp() {
 struct timeval tv ;
 gettimeofday( &tv, NULL ) ;
 return (double) tv.tv_usec/1000 + tv.tv_sec*1000 ;
}
void initData(float* data, int n){ 
 int i,j,k;
 for(i = 0; i < n; i++){
  for(j = 0; j < n; j++){
   for(k = 0; k < n; k++){
    data[i*n*n + j*n + k] = (float) (i+j+k)*1.1;
   }
  }
 }
}
void debugPrint(float* data, int n){
 int i,j,k;
 for(i = 0; i < 3; i++){
  printf("--------layer %d--------\n",i);
  for(j = 0; j < n; j++){
   for(k = 0; k < n; k++){
    printf("%lf ",data[i*n*n + j*n + k]);
   }
   printf("\n");
  }
  printf("\n");
 }
 printf("\n");
}
// host side matrix addition
void h_stencil(float *a, float *b, int n){
 int i,j,k;
 for(i = 1; i < n-1; i++){
  for(j = 1; j < n-1; j++){
   for(k = 1; k < n-1; k++){
    a[i*n*n + j*n + k] = 0.8*(b[(i-1)*n*n+j*n+k]+b[(i+1)*n*n+j*n+k]+b[i*n*n+(j-1)*n+k]+b[i*n*n+(j+1)*n+k]+b[i*n*n+j*n+(k-1)]+b[i*n*n+j*n+(k+1)]);
   }
  }
 }
}
// host side validation 
bool val(float *a, float *b, int n){
 int i,j,k;
 bool match = true;
 for(i = 0; i < n; i++){
  for(j = 0; j < n; j++){
   for(k = 0; k < n; k++){
    if(match && (round(a[i*n*n + j*n + k]*100)/100 != round(b[i*n*n+j*n+k]*100)/100)){
     //printf("%d,%d,%d expect %lf, actual %lf\n",i,j,k,h_A[i*n*n + j*n + k],h_dA[i*n*n+j*n+k]);
     match = false;
     //break;
    }
   }
  }
 }
 return match;
}
float h_sum(float *data, int n){
 int i,j,k;
 float ret=0;
 for(i = 0; i < n; i++){
  for(j = 0; j < n; j++){
   for(k = 0; k < n; k++){
    ret += data[i*n*n + j*n + k]*(((i+j+k)%2)?1:-1);
   }
  }
 }
 return ret;
}
__device__ void globalToShared(float *sm, float *b, int l, int n, int smx, int smy, int ix, int iy){
  sm[smx+smy*(blockDim.x+2)] = b[ix + iy*n + l*n*n];
  if(smx==1){
   sm[0+smy*(blockDim.x+2)] = b[ix-1 + iy*n + l*n*n];
  }
  if(smx==blockDim.x || ix==n-2){
   sm[smx+1+smy*(blockDim.x+2)] = b[ix+1 + iy*n + l*n*n];
  }
  if(smy==1){
   sm[smx] = b[ix + (iy-1)*n + l*n*n];
  }
  if(smy==blockDim.y || iy==n-2){
   sm[smx+(smy+1)*(blockDim.x+2)] = b[ix + (iy+1)*n + l*n*n];
  }
}
__global__ void kernal( float *a, float *b, int n){
 extern __shared__ float sm[];
 int ix = threadIdx.x + 1;
 int iy = threadIdx.y + 1;
 int gx = threadIdx.x + 1 + blockIdx.x*blockDim.x;
 int gy = threadIdx.y + 1 + blockIdx.y*blockDim.y;
 float down,up,self;
 float l1;
 if(gx<n-1&&gy<n-1){
  globalToShared(sm, b, 0, n, ix, iy, gx, gy);
  __syncthreads();
  down = sm[ix + iy*(blockDim.x+2)];
  globalToShared(sm, b, 1, n, ix, iy, gx, gy);
  __syncthreads();
  self = sm[ix + iy*(blockDim.x+2)];
  l1 = sm[ix-1 + iy*(blockDim.x+2)] + sm[ix+1 + iy*(blockDim.x+2)] + sm[ix + (iy-1)*(blockDim.x+2)] + sm[ix + (iy+1)*(blockDim.x+2)];
  __syncthreads();
  int layer;
  #pragma unroll
  for(layer = 2; layer < n; layer++){
   globalToShared(sm, b, layer, n, ix, iy, gx, gy);
   __syncthreads();
   up = sm[ix + iy*(blockDim.x+2)];
   a[gx + gy*n + (layer-1)*n*n] = 0.8*(down+up+l1);
   down = self;
   self = up;
   l1 = sm[ix-1 + iy*(blockDim.x+2)] + sm[ix+1 + iy*(blockDim.x+2)] + sm[ix + (iy-1)*(blockDim.x+2)] + sm[ix + (iy+1)*(blockDim.x+2)];
   __syncthreads();
  }
 }
}
int main( int argc, char *argv[] ) {
 // get program arguments
 if( argc != 2) {
 printf("Error: wrong number of args\n") ;
 exit(1) ;
 }
 int n = atoi( argv[1] );
 int noElems = n*n*n ;
 int bytes = noElems * sizeof(float) ;

 // alloc memory host-side
 float *h_A = (float *) malloc( bytes ) ;
 float *h_B = (float *) malloc( bytes ) ;
 float *h_dA = (float *) malloc( bytes ) ;

 // init matrices with random data
 initData(h_B, n);
 memset(h_A, 0, bytes);
 
 // alloc memory dev-side
 float *d_A, *d_B ;
 hipMalloc( (void **) &d_A, bytes ) ;
 hipMalloc( (void **) &d_B, bytes ) ;

 double timeStampA = getTimeStamp() ;
 //transfer data to dev
 hipMemcpy( d_B, h_B, bytes, hipMemcpyHostToDevice ) ;
 double timeStampB = getTimeStamp() ;

 //debugPrint(h_B, n);
 // invoke Kernel
 dim3 block(32, 32);
 dim3 grid((n-2+block.x-1)/block.x,(n-2+block.y-1)/block.y);
 kernal<<<grid,block,(1024+33*4)*sizeof(float)>>>(d_A,d_B,n);
 hipDeviceSynchronize() ;
 //cudaDeviceProp GPUprop;
 //cudaGetDeviceProperties(&GPUprop,0);
 //printf("maxgridsize x is %d\n",GPUprop.maxGridSize[0]);

 double timeStampC = getTimeStamp() ;
 //copy data back
 hipMemcpy( h_dA, d_A, bytes, hipMemcpyDeviceToHost ) ;
 double timeStampD = getTimeStamp() ;

 h_stencil(h_A,h_B,n);
 //float h_Result = h_sum(h_A,n);
 float h_dResult = h_sum(h_dA,n);
 
 // print out results
 //if(!memcmp(h_A,h_dA,n*n*n*sizeof(float))){
 if(val(h_A,h_dA,n)){
  //debugPrint(h_hC, nx, ny);
  //debugPrint(h_dC, nx, ny);
  FILE* fptr;
  fptr = fopen("time.log","a");
  fprintf(fptr,"%d: %lf, %.6f %.6f %.6f %.6f\n", n, h_dResult, timeStampD-timeStampA, timeStampB-timeStampA, timeStampC-timeStampB, timeStampD-timeStampC);
  fclose(fptr);
  printf("%lf %d\n", h_dResult, (int)round(timeStampD-timeStampA));
 }else{
  //debugPrint(h_A, n);
  //debugPrint(h_dA, n);
  printf("Error: function failed.\n");
 }
 
 // free GPU resources
 hipFree(d_A);
 hipFree(d_B);
 hipDeviceReset();
}
